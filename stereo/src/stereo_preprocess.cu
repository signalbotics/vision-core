#include "hip/hip_runtime.h"
#include "stereo_preprocess.h"
#include <opencv2/opencv.hpp>

__global__ void stereo_preprocess_kernel(uint8_t* src, int src_line_size, int src_width, int src_height, float* dst, int edge)
{
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= edge) 
        return;

    int dx = position % src_width;
    int dy = position / src_width;

    float c0, c1, c2;

    uint8_t* v = src + dy * src_line_size + dx * 3;

    c0 = v[0];
    c1 = v[1];
    c2 = v[2];

    //bgr to rgb 
    float t = c2;
    c2 = c0;
    c0 = t;

    //normalization
    // c0 = (c0 / 255.0f - 0.485) / 0.229;
    // c1 = (c1 / 255.0f - 0.456) / 0.224;
    // c2 = (c2 / 255.0f - 0.406) / 0.225;

    //rgbrgbrgb to rrrgggbbb
    int area = src_width * src_height;
    float* pdst_c0 = dst + dy * src_width + dx;
    float* pdst_c1 = pdst_c0 + area;
    float* pdst_c2 = pdst_c1 + area;
    *pdst_c0 = c0;
    *pdst_c1 = c1;
    *pdst_c2 = c2;
}

__global__ void stereo_reprojectImageTo3D_kernel(uint8_t* left_img,float*disparity,float*pointcloud,float*Q_device,int disparity_rows,int disparity_cols)
{
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    if(tid>=disparity_cols*disparity_rows)
    {
        return;
    }
    int col=tid%disparity_cols;
    int row=tid/disparity_cols;

    uint8_t* v = left_img + row * disparity_cols*3 + col * 3;

    //Q*disparity
    float w=Q_device[14]*disparity[row*disparity_cols+col];

    for (size_t i = 0; i < 3; i++)
    {
        pointcloud[(row*disparity_cols+col)*6+i]=(Q_device[i*4]*col+Q_device[i*4+1]*row+Q_device[i*4+3]*1)/w;  
    }
    pointcloud[(row*disparity_cols+col)*6+3]=(float)v[2]; 
    pointcloud[(row*disparity_cols+col)*6+4]=(float)v[1];
    pointcloud[(row*disparity_cols+col)*6+5]=(float)v[0];
}



void stereo_preprocess(uint8_t* src, float* dst, int src_width, int src_height, hipStream_t stream) 
{

    int jobs = src_width * src_height;
    int threads = 256;
    int blocks = (jobs +threads-1)/threads;
    stereo_preprocess_kernel<<<blocks, threads, 0, stream>>>(src, src_width*3, src_width, src_height, dst, jobs);
}

void stereo_reprojectImageTo3D(uint8_t* left_img,float*disparity,float*pointcloud,float*Q_device,int disparity_rows,int disparity_cols)
{
    int jobs=disparity_rows*disparity_cols;
    int threads=256;
    int blocks=(jobs+threads-1)/threads;
    stereo_reprojectImageTo3D_kernel<<<blocks,threads>>>(left_img,disparity,pointcloud,Q_device,disparity_rows,disparity_cols);
}